#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA kernel, each thread takes care of one element of c
__global__ void vec_addition(double *a, double *b, double *c, int n) {
  //get the global thread ID
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  //be sure that id is less than n
  if (id < n)
    c[id] = a[id] + b[id];
}

int main(int argc, char* argv[]) {
  int n = 10, i;

  //host io vectors
  double *h_a;
  double *h_b;
  double *h_c;

  //device io vectors
  double *d_a;
  double *d_b;
  double *d_c;

  size_t bytes = n * sizeof(double);

  //allocating memory for each vector on host
  h_a = (double *)malloc(bytes);
  h_b = (double *)malloc(bytes);
  h_c = (double *)malloc(bytes);

  //allocating memory for each vector on GPU
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  for (i = 0; i < n; i++) {
    h_a[i] = i + 1;
    h_b[i] = n - i;
  }

  //copy host vectors to device
  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  int blockSize, gridSize;

  //number of threads in each thread block
  blockSize = 256;

  //number of thread blocks in grid
  gridSize = (int)ceil((float)n / blockSize);

  //executing the kernel
  vec_addition<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

  //copy array solution to host
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  int sum = 0;
  for (int i = 0, i < n; i++) sum += h_c[i];

  printf("the result is: %d\n", sum);

  //release device memory this is very important
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  //release host memory
  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}
